#include "hip/hip_runtime.h"
#include <Hornet.hpp>
#include "StandardAPI.hpp"
#include "Util/BatchFunctions.hpp"
#include "Util/RandomGraphData.cuh"
#include <Host/FileUtil.hpp>            //xlib::extract_filepath_noextension
#include <Device/Util/CudaUtil.cuh>     //xlib::deviceInfo
#include <algorithm>                    //std:.generate
#include <chrono>                       //std::chrono
#include <random>                       //std::mt19937_64
#include <hip/hip_runtime_api.h>
#include <Core/Static/Static.cuh>
#include <omp.h>

//using namespace hornets_nest;
using namespace timer;
using namespace std::string_literals;

using vert_t = int;
using eoff_t = int;
using HornetGPU = hornet::gpu::Hornet<vert_t>;
using UpdatePtr = hornet::BatchUpdatePtr<vert_t, hornet::EMPTY, hornet::DeviceType::HOST>;
using Update = hornet::gpu::BatchUpdate<vert_t>;
using Init = hornet::HornetInit<vert_t>;
using hornet::SoAData;
using hornet::TypeList;
using hornet::DeviceType;

/**
 * @brief Example tester for Hornet
 */
int exec(int argc, char* argv[]) {
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    graph::GraphStd<vert_t, vert_t> graph;
    graph.read(argv[1]);
    int batch_size = std::stoi(argv[2]);
    Init hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(), graph.csr_out_edges());

    bool err = false;
    omp_set_num_threads(2);
#pragma omp parallel
  {
    hipSetDevice(omp_get_thread_num());
    HornetGPU hornet_gpu;
    hornet_gpu.reset(hornet_init);
    auto init_coo = hornet_gpu.getCOO(true);

    hornet::RandomGenTraits<hornet::EMPTY> cooGenTraits;
    auto randomBatch = hornet::generateRandomCOO<vert_t, eoff_t>(graph.nV(), batch_size, cooGenTraits);
    Update batch_update(randomBatch);
    hornet_gpu.insert(batch_update);
    auto inst_coo = hornet_gpu.getCOO(true);
    init_coo.append(randomBatch);
    init_coo.sort();

    hornet::COO<DeviceType::HOST, vert_t, hornet::EMPTY, eoff_t> host_init_coo = init_coo;
    hornet::COO<DeviceType::HOST, vert_t, hornet::EMPTY, eoff_t> host_inst_coo = inst_coo;

    auto *s = host_init_coo.srcPtr();
    auto *d = host_init_coo.dstPtr();
    auto *S = host_inst_coo.srcPtr();
    auto *D = host_inst_coo.dstPtr();
    auto len = host_init_coo.size();
    if (host_inst_coo.size() != host_init_coo.size()) {
      std::cerr<<"\nInit Size "<<host_init_coo.size()<<" != Combined size "<<host_inst_coo.size()<<"\n";
      len = std::min(host_init_coo.size(), host_inst_coo.size());
    }
    for (int i = 0; i < len; ++i) {
      if ((s[i] != S[i]) || (d[i] != D[i])) {
        err = true;
        std::cout<<"ERR : ";
        std::cout<<s[i]<<" "<<d[i]<<"\t";
        std::cout<<"\t\t";
        std::cout<<S[i]<<" "<<D[i];
        std::cout<<"\n";
      }
    }
  }
    if (!err) {
      std::cout<<"PASSED\n";
    } else {
      std::cout<<"NOT PASSED\n";
    }

    return 0;
}

int main(int argc, char* argv[]) {
    int ret = 0;
#if defined(RMM_WRAPPER)
    gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
#endif

    ret = exec(argc, argv);

#if defined(RMM_WRAPPER)
    }//scoping technique to make sure that gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    gpu::finalizeRMMPoolAllocation();
#endif

    return ret;
}

